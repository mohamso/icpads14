#include "Sync.h"

#define checkCuda(error) __checkCuda(error, __FILE__, __LINE__)

////////////////////////////////////////////////////////////////////////////////
// A method for checking error in CUDA calls
////////////////////////////////////////////////////////////////////////////////
inline void __checkCuda(hipError_t error, const char *file, const int line)
{
	#if defined(DEBUG) || defined(_DEBUG)
	if (error != hipSuccess)
	{
		printf("checkCuda error at %s:%i: %s\n", file, line, hipGetErrorString(hipGetLastError()));
		exit(-1);
	}
	#endif

	return;
}

///////////////////////////////////////////////////
// Function that scans for devices on a single node
///////////////////////////////////////////////////
extern "C" int DeviceScan()
{
	int numberOfDevices;
	checkCuda(hipGetDeviceCount(&numberOfDevices));

	return numberOfDevices;
}

//////////////////////////////////////////////////////////////////
// Function that checks if multiple GPUs are available on the node
//////////////////////////////////////////////////////////////////
extern "C" void MPIDeviceCheck(int rank, int numberOfProcesses, int numberOfDevices)
{
	if (numberOfDevices < 2)
	{
		printf("Less than two devices were found.\n");
		printf("Exiting...\n");
		Finalize();
		exit(-1);

	}

	if (numberOfProcesses > numberOfDevices)
	{
		printf("Number of processors exceeds the number of GPUs\n");
		printf("Exiting...\n");
		Finalize();
		exit(-1);
	}
}

////////////////////////////////////////////////////////////////
// Function that assigns a single device to a single MPI process
////////////////////////////////////////////////////////////////
extern "C" void AssignDevices(int rank)
{
	int numberOfDevices = 0;

	checkCuda(hipGetDeviceCount(&numberOfDevices));
	checkCuda(hipSetDevice(rank % numberOfDevices));

	printf("Process %d -> GPU%d\n", rank, rank % numberOfDevices);
}

///////////////////////////////////////////////////////////////////////
// Function that checks if ECC is turned on for the devices on the node
///////////////////////////////////////////////////////////////////////
extern "C" void ECCCheck(int rank)
{
	hipDeviceProp_t properties;

    checkCuda(hipGetDeviceProperties(&properties, rank));

    if (properties.ECCEnabled == 1)
    {
        printf("ECC is turned on for device #%d\n", rank);
    }
    else
    {
        printf("ECC is turned off for device #%d\n", rank);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Computes the thread block size
////////////////////////////////////////////////////////////////////////////////
extern "C" int getBlock(int n, int block)
{
	return (n+2)/block + ((n+2)%block == 0?0:1);
}
